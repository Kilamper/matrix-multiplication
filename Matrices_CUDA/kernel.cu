﻿/*
===========================================================================
Nombre del proyecto:    Matrices_CUDA
Autor:                  Kilian Armas Pérez
Titulación:             Ingeniería Informática
Asignatura:             Arquitectura de Computadores
Fecha:                  09/07/2023
Descripción:            Implementación del algoritmo de multiplicación de
                        matrices con números en coma flotante haciendo uso
                        de la librería de programación paralela CUDA.
===========================================================================
*/

#include "hip/hip_runtime.h"

#include <iostream>
#include <cstdlib>
#include <chrono>

// Kernel de multiplicación de matrices en CUDA
__global__ void multiplicar_matrices(double* a, double* b, double* c, int N, int M, int P) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < P) {
        double value = 0.0;
        for (int k = 0; k < M; ++k) {
            value += a[row * M + k] * b[k * P + col];
        }
        c[row * P + col] = value;
    }
}

int main() {
    const int N = 3000; // Filas de la matriz A y C
    const int M = 3000; // Columnas de la matriz A y filas de la matriz B
    const int P = 3000; // Columnas de la matriz B y C

    puts("\nMatriz x Matriz Paralelo - CUDA - BEGIN\n");

    // Definir las matrices de entrada y salida
    double* a = new double[N * M];
    double* b = new double[M * P];
    double* c = new double[N * P];

    // Inicializar las matrices de entrada
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < M; j++) {
            a[i * M + j] = static_cast<double>(j);
        }
    }
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < P; j++) {
            b[i * P + j] = static_cast<double>(j);
        }
    }

    // Declarar punteros de dispositivo CUDA
    double* dev_a;
    double* dev_b;
    double* dev_c;

    // Iniciar el contador de tiempo
    std::chrono::high_resolution_clock::time_point start = std::chrono::high_resolution_clock::now();

    // Alojar memoria en el dispositivo CUDA
    hipMalloc((void**)&dev_a, N * M * sizeof(double));
    hipMalloc((void**)&dev_b, M * P * sizeof(double));
    hipMalloc((void**)&dev_c, N * P * sizeof(double));

    // Copiar los datos de las matrices de entrada al dispositivo CUDA
    hipMemcpy(dev_a, a, N * M * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, M * P * sizeof(double), hipMemcpyHostToDevice);

    // Configurar la cuadrícula y el bloque de hilos para la ejecución del kernel
    dim3 blockSize(16, 16);
    dim3 gridSize((P + blockSize.x - 1) / blockSize.x, (N + blockSize.y - 1) / blockSize.y);

    // Ejecutar el kernel de multiplicación de matrices en CUDA
    multiplicar_matrices<<<gridSize, blockSize>>>(dev_a, dev_b, dev_c, N, M, P);

    // Copiar el resultado de vuelta desde el dispositivo CUDA a la memoria del host
    hipMemcpy(c, dev_c, N * P * sizeof(double), hipMemcpyDeviceToHost);

    // Detener el contador de tiempo y calcular la duración
    std::chrono::high_resolution_clock::time_point end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> duration = std::chrono::duration_cast<std::chrono::duration<double>>(end - start);

    // Imprimir las matrices y el resultado
    std::cout << "Matriz A (" << N << " x " << M << "):" << std::endl;
    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < M; ++j) {
            std::cout << a[i * M + j] << " ";
        }
        std::cout << std::endl;
        if (N > 16 && i == 1) {
            std::cout << "..." << std::endl;
            break;
        }
    }

    std::cout << std::endl << "Matriz B(" << M << " x " << P << "):" << std::endl;
    for (int i = 0; i < M; ++i) {
        for (int j = 0; j < P; ++j) {
            std::cout << b[i * P + j] << " ";
        }
        std::cout << std::endl;
        if (M > 16 && i == 1) {
            std::cout << "..." << std::endl;
            break;
        }
    }

    std::cout << std::endl << "Matriz Resultado C (" << N << " x " << P << "):" << std::endl;
    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < P; ++j) {
            std::cout << c[i * P + j] << " ";
        }
        std::cout << std::endl;
        if (M > 16 && i == 1) {
            std::cout << "..." << std::endl;
            break;
        }
    }
    std::cout << std::endl << "Tiempo de ejecucion: " << duration.count() * 1000 << " milisegundos" << std::endl;

    puts("\nMatriz x Matriz Paralelo - CUDA - FINISH\n");

    // Liberar memoria en el dispositivo CUDA
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    // Liberar memoria en el host
    delete[] a;
    delete[] b;
    delete[] c;

    return 0;
}